#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <math.h>
#include <stdint.h>
#include <inttypes.h>

//#include <iostream>
extern "C" {
#include "defs.h"
//#include "function_precision.h"
//#include "utils.h"
//#include "gridlink_utils_double.h"

//#include "weight_functions_double.h"

#include "cellarray_mocks_double.h"
#include "cellarray_mocks_float.h"

#include "countpairs_s_mu_mocks_gpu.h"
#include <hip/hip_runtime.h>

// Define pair_struct_double here instead of including weight_functions_double
// Info about a particle pair that we will pass to the weight function
typedef struct
{
    double weights0[MAX_NUM_WEIGHTS];
    double weights1[MAX_NUM_WEIGHTS];
    double dx, dy, dz;

    // These will only be present for mock catalogs
    double parx, pary, parz;

    // Add for angular weights
    double costheta;

    double *p_weight;
    double *p_sep;
    int p_num;
    //pair_weight_struct_double pair_weight;

    int num_weights;
    uint8_t num_integer_weights;
    int8_t noffset;
    double default_value;
} pair_struct_double;

typedef struct
{
    float weights0[MAX_NUM_WEIGHTS];
    float weights1[MAX_NUM_WEIGHTS];
    float dx, dy, dz;

    // These will only be present for mock catalogs
    float parx, pary, parz;

    // Add for angular weights
    float costheta;

    float *p_weight;
    float *p_sep;
    int p_num;
    //pair_weight_struct_float pair_weight;

    int num_weights;
    uint8_t num_integer_weights;
    int8_t noffset;
    float default_value;
} pair_struct_float;

}

//device function to do inverse_bitwise weighting
__device__ double inverse_bitwise_double(pair_struct_double *pair){
    int nbits = pair->noffset;
    for (int w=0;w<pair->num_integer_weights;w++) {
        nbits += __popc(*((long *) &(pair->weights0[w])) & *((long *) &(pair->weights1[w])));
    }
    double weight = (nbits == 0) ? pair->default_value : 1./nbits;
    int num = pair->p_num;
    if (num) {
        double costheta = pair->costheta;
        double *pair_sep = pair->p_sep;
        if (costheta > pair_sep[num-1] || (costheta <= pair_sep[0])) {
            ;
        }
        else {
            double *pair_weight = pair->p_weight;
            for (int kbin=0;kbin<num-1;kbin++) {
                if(costheta <= pair_sep[kbin+1]) { // ]min, max], as costheta instead of theta
                    double frac = (costheta - pair_sep[kbin])/(pair_sep[kbin+1] - pair_sep[kbin]);
                    weight *= (1 - frac) * pair_weight[kbin] + frac * pair_weight[kbin+1];
                    break;
                }
            }
        }
    }
    num = pair->num_weights;
    int numi = pair->num_integer_weights;
    if (num > numi) weight *= pair->weights0[numi]*pair->weights1[numi]; // multiply by the first float weight
    numi++;
    if (num > numi) weight -= pair->weights0[numi]*pair->weights1[numi]; // subtract the second float weight
    return weight;
}

__device__ float inverse_bitwise_float(pair_struct_float *pair){
    int nbits = pair->noffset;
    for (int w=0;w<pair->num_integer_weights;w++) {
        nbits += __popc(*((long *) &(pair->weights0[w])) & *((long *) &(pair->weights1[w])));
    }
    float weight = (nbits == 0) ? pair->default_value : 1./nbits;
    int num = pair->p_num;
    if (num) {
        float costheta = pair->costheta;
        float *pair_sep = pair->p_sep;
        if (costheta > pair_sep[num-1] || (costheta <= pair_sep[0])) {
            ;
        }
        else {
            float *pair_weight = pair->p_weight;
            for (int kbin=0;kbin<num-1;kbin++) {
                if(costheta <= pair_sep[kbin+1]) { // ]min, max], as costheta instead of theta
                    float frac = (costheta - pair_sep[kbin])/(pair_sep[kbin+1] - pair_sep[kbin]);
                    weight *= (1 - frac) * pair_weight[kbin] + frac * pair_weight[kbin+1];
                    break;
                }
            }
        }
    }
    num = pair->num_weights;
    int numi = pair->num_integer_weights;
    if (num > numi) weight *= pair->weights0[numi]*pair->weights1[numi]; // multiply by the first float weight
    numi++;
    if (num > numi) weight -= pair->weights0[numi]*pair->weights1[numi]; // subtract the second float weight
    return weight;
}


__global__ void countpairs_s_mu_mocks_kernel_double(double *x0, double *y0, double *z0,
               double *x1, double *y1, double *z1, int N,
               int *np0, int *np1, 
               int *same_cell, int64_t *icell0, int64_t *icell1, 
               int *cellpair_lut, int *cellthread_lut,
               int *start_idx0, int *start_idx1,
               double *min_xdiff, double *min_ydiff, 
               double *savg, int *npairs, const double *supp_sqr,
               const double sqr_smax, const double sqr_smin, const int nsbin,
               const int nmu_bins, 
               const double sqr_mumax, const double inv_dmu, const double mumin_invstep,
               double inv_sstep, double smin_invstep, const selection_struct selection,
               int need_savg, int autocorr, int los_type, int bin_type) {
    //thread index tidx 
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= N) return;

    int icellpair = cellpair_lut[blockIdx.x]; //use block index to find cellpair index
    int cell_tidx = cellthread_lut[blockIdx.x] + threadIdx.x; //index within this cellpair from 0 to np0*np1-1

    //icell0, icell1 will translate icellpair to a cell within each lattice
    //start_idx0, start_idx1 then translate to i, j in x0,y0,z0 and x1,y1,z1
    int64_t cellindex0 = icell0[icellpair];
    int64_t cellindex1 = icell1[icellpair];
    //nthreads = np0 * np1 for each cell pair icell
    int this_np0 = np0[cellindex0];
    int this_np1 = np1[cellindex1];
    if (cell_tidx >= this_np0*this_np1) return;

    //start_idx0, start_idx1 give index for first element of x0, y0, z0 and x1, y1, z1 in cell icell.
    //% and / to get i, j
    int i = start_idx0[cellindex0] + cell_tidx / this_np1;
    int j = start_idx1[cellindex1] + cell_tidx % this_np1;

    //get positions for each particle
    double xpos = x0[i];
    double ypos = y0[i];
    double zpos = z0[i];

    double x1pos = x1[j];
    double y1pos = y1[j];
    double z1pos = z1[j];

    if (same_cell[icellpair] && z1pos <= zpos) { 
        //return if same particle or in same cell with z1 < z0
        //this way we do not double count pairs
        return;
    }

    const double max_dz = sqrt(sqr_smax - min_xdiff[icellpair]*min_xdiff[icellpair] - min_ydiff[icellpair]*min_ydiff[icellpair]);
    const double this_dz = z1pos-zpos;
    if (abs(this_dz) >= max_dz) {
        //particle too far away in z
        return;
    }

    const double this_dx = x1pos-xpos;
    const double this_dy = y1pos-ypos;
    if ((this_dx*this_dx + this_dy*this_dy + this_dz*this_dz) >= sqr_smax) {
        //particle too far away in separation
        return;
    } 

    //hat1 calcs are done if need_weightavg || ((autocorr == 1) && (los_type == FIRSTPOINT_LOS)
    //need_weightavg is FALSE by definition in this kernel so remove that part of conditional 
    double xhat1=NULL, yhat1=NULL, zhat1=NULL;
    if (((autocorr == 1) && (los_type == FIRSTPOINT_LOS))) {
        const double norm1 = sqrt(x1pos*x1pos + y1pos*y1pos + z1pos*z1pos);
        xhat1 = x1pos/norm1;
        yhat1 = y1pos/norm1;
        zhat1 = z1pos/norm1;
    }

    //hat1 calcs are done if need_weightavg || los_type == FIRSTPOINT_LOS 
    //need_weightavg is FALSE by definition in this kernel so remove that part of conditional 
    double xhat0=NULL, yhat0=NULL, zhat0=NULL;
    if (los_type == FIRSTPOINT_LOS) {
        const double norm0 = sqrt(xpos*xpos + ypos*ypos + zpos*zpos);
        xhat0 = xpos/norm0;
        yhat0 = ypos/norm0;
        zhat0 = zpos/norm0;
    }

    const double parx = xpos + x1pos;
    const double pary = ypos + y1pos; 
    const double parz = zpos + z1pos; 

    const double perpx = x1pos - xpos;
    const double perpy = y1pos - ypos;
    const double perpz = z1pos - zpos;

    //return if greater than max dz
    if (perpz > max_dz) {
        return;
    }

    const double sqr_s = perpx*perpx + perpy*perpy + perpz*perpz;
    if(sqr_s >= sqr_smax || sqr_s < sqr_smin) {
        return;
    } 

    double s = 0;
    int mubin = nmu_bins, mubin2 = nmu_bins;
    if (sqr_s <= 0.) {
        mubin2 = mubin = (int) mumin_invstep;
        if((selection.selection_type & RP_SELECTION) && ((0. < selection.rpmin_sqr) || (0. >= selection.rpmax_sqr))) return;
    } 
    else if (los_type == MIDPOINT_LOS) {
        const double s_dot_l = parx*perpx + pary*perpy + parz*perpz;
        const double sqr_l = parx*parx + pary*pary + parz*parz;
        const double sqr_mu = s_dot_l * s_dot_l / (sqr_l * sqr_s);
        if (sqr_mu >= sqr_mumax) {
            return;
        } 
        if (selection.selection_type & RP_SELECTION) {
            const double sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) return;
        }
        const double mu = s_dot_l >= 0 ? sqrt(sqr_mu) : -sqrt(sqr_mu);
        mubin = (int) (mu * inv_dmu + mumin_invstep);
        if (autocorr == 1) mubin2 = (int) (- mu * inv_dmu + mumin_invstep);
        if(need_savg || bin_type == BIN_LIN) s = sqrt(sqr_s);
    }
    else {
        const double s_dot_l = xhat0*perpx + yhat0*perpy + zhat0*perpz;
        const double sqr_mu = s_dot_l * s_dot_l / sqr_s;

        int skip_mu = (sqr_mu >= sqr_mumax);
        if (selection.selection_type & RP_SELECTION) {
            const double sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu = 1;
        }
        if (autocorr == 1) {
            const double s_dot_l2 = xhat1*perpx + yhat1*perpy + zhat1*perpz;
            const double sqr_mu2 = s_dot_l2 * s_dot_l2 / sqr_s;
            int skip_mu2 = (sqr_mu2 >= sqr_mumax);
            if (selection.selection_type & RP_SELECTION) {
                const double sqr_rp = (1. - sqr_mu2) * sqr_s; 
                if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu2 = 1;
            }
            if (skip_mu && skip_mu2) {
                return;
            }
            s = sqrt(sqr_s);
            if (skip_mu == 0) mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
            if (skip_mu2 == 0) mubin2 = (int) (- s_dot_l2 / s * inv_dmu + mumin_invstep);
        }
        else {
            if (skip_mu) {
                return;
            }
            s = sqrt(sqr_s);
            mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
	}
    }

    double sw = s;
    int kbin = 0;
    if (bin_type == BIN_LIN) {
	kbin = (int) (s*inv_sstep + smin_invstep);
    }
    else {
	for(kbin=nsbin-1;kbin>=1;kbin--) {
	    if(sqr_s >= supp_sqr[kbin-1]) {
		break;
	    }
	}//finding kbin
    }
    kbin *= nmu_bins+1;
    {
	const int ibin = kbin + mubin;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
    }
    if (autocorr == 1) {
	const int ibin = kbin + mubin2;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
    }
}

__global__ void countpairs_s_mu_mocks_kernel_float(float *x0, float *y0, float *z0,
               float *x1, float *y1, float *z1, int N,
               int *np0, int *np1, 
               int *same_cell, int64_t *icell0, int64_t *icell1, 
               int *cellpair_lut, int *cellthread_lut,
               int *start_idx0, int *start_idx1,
               float *min_xdiff, float *min_ydiff, 
               float *savg, int *npairs, const float *supp_sqr,
               const float sqr_smax, const float sqr_smin, const int nsbin,
               const int nmu_bins, 
               const float sqr_mumax, const float inv_dmu, const float mumin_invstep,
               float inv_sstep, float smin_invstep, const selection_struct selection,
               int need_savg, int autocorr, int los_type, int bin_type) {
    //thread index tidx 
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= N) return;

    int icellpair = cellpair_lut[blockIdx.x]; //use block index to find cellpair index
    int cell_tidx = cellthread_lut[blockIdx.x] + threadIdx.x; //index within this cellpair from 0 to np0*np1-1

    //icell0, icell1 will translate icellpair to a cell within each lattice
    //start_idx0, start_idx1 then translate to i, j in x0,y0,z0 and x1,y1,z1
    int64_t cellindex0 = icell0[icellpair];
    int64_t cellindex1 = icell1[icellpair];
    //nthreads = np0 * np1 for each cell pair icell
    int this_np0 = np0[cellindex0];
    int this_np1 = np1[cellindex1];
    if (cell_tidx >= this_np0*this_np1) return;

    //start_idx0, start_idx1 give index for first element of x0, y0, z0 and x1, y1, z1 in cell icell.
    //% and / to get i, j
    int i = start_idx0[cellindex0] + cell_tidx / this_np1;
    int j = start_idx1[cellindex1] + cell_tidx % this_np1;

    //get positions for each particle
    float xpos = x0[i];
    float ypos = y0[i];
    float zpos = z0[i];

    float x1pos = x1[j];
    float y1pos = y1[j];
    float z1pos = z1[j];

    if (same_cell[icellpair] && z1pos <= zpos) { 
        //return if same particle or in same cell with z1 < z0
        //this way we do not float count pairs
        if (z1pos < zpos) return;
        if (z1pos == zpos && j <= i) return;
        //return;
    }

    const float max_dz = sqrt(sqr_smax - min_xdiff[icellpair]*min_xdiff[icellpair] - min_ydiff[icellpair]*min_ydiff[icellpair]);
    const float this_dz = z1pos-zpos;
    if (abs(this_dz) >= max_dz) {
        //particle too far away in z
        return;
    }

    const float this_dx = x1pos-xpos;
    const float this_dy = y1pos-ypos;
    if ((this_dx*this_dx + this_dy*this_dy + this_dz*this_dz) >= sqr_smax) {
        //particle too far away in separation
        return;
    } 

    //hat1 calcs are done if need_weightavg || ((autocorr == 1) && (los_type == FIRSTPOINT_LOS)
    //need_weightavg is FALSE by definition in this kernel so remove that part of conditional 
    float xhat1=NULL, yhat1=NULL, zhat1=NULL;
    if (((autocorr == 1) && (los_type == FIRSTPOINT_LOS))) {
        const float norm1 = sqrt(x1pos*x1pos + y1pos*y1pos + z1pos*z1pos);
        xhat1 = x1pos/norm1;
        yhat1 = y1pos/norm1;
        zhat1 = z1pos/norm1;
    }

    //hat1 calcs are done if need_weightavg || los_type == FIRSTPOINT_LOS 
    //need_weightavg is FALSE by definition in this kernel so remove that part of conditional 
    float xhat0=NULL, yhat0=NULL, zhat0=NULL;
    if (los_type == FIRSTPOINT_LOS) {
        const float norm0 = sqrt(xpos*xpos + ypos*ypos + zpos*zpos);
        xhat0 = xpos/norm0;
        yhat0 = ypos/norm0;
        zhat0 = zpos/norm0;
    }

    const float parx = xpos + x1pos;
    const float pary = ypos + y1pos; 
    const float parz = zpos + z1pos; 

    const float perpx = x1pos - xpos;
    const float perpy = y1pos - ypos;
    const float perpz = z1pos - zpos;

    //return if greater than max dz
    if (perpz > max_dz) {
        return;
    }

    const float sqr_s = perpx*perpx + perpy*perpy + perpz*perpz;
    if(sqr_s >= sqr_smax || sqr_s < sqr_smin) {
        return;
    } 

    float s = 0;
    int mubin = nmu_bins, mubin2 = nmu_bins;
    if (sqr_s <= 0.) {
        mubin2 = mubin = (int) mumin_invstep;
        if((selection.selection_type & RP_SELECTION) && ((0. < selection.rpmin_sqr) || (0. >= selection.rpmax_sqr))) return;
    } 
    else if (los_type == MIDPOINT_LOS) {
        const float s_dot_l = parx*perpx + pary*perpy + parz*perpz;
        const float sqr_l = parx*parx + pary*pary + parz*parz;
        const float sqr_mu = s_dot_l * s_dot_l / (sqr_l * sqr_s);
        if (sqr_mu >= sqr_mumax) {
            return;
        } 
        if (selection.selection_type & RP_SELECTION) {
            const float sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) return;
        }
        const float mu = s_dot_l >= 0 ? sqrt(sqr_mu) : -sqrt(sqr_mu);
        mubin = (int) (mu * inv_dmu + mumin_invstep);
        if (autocorr == 1) mubin2 = (int) (- mu * inv_dmu + mumin_invstep);
        if(need_savg || bin_type == BIN_LIN) s = sqrt(sqr_s);
    }
    else {
        const float s_dot_l = xhat0*perpx + yhat0*perpy + zhat0*perpz;
        const float sqr_mu = s_dot_l * s_dot_l / sqr_s;

        int skip_mu = (sqr_mu >= sqr_mumax);
        if (selection.selection_type & RP_SELECTION) {
            const float sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu = 1;
        }
        if (autocorr == 1) {
            const float s_dot_l2 = xhat1*perpx + yhat1*perpy + zhat1*perpz;
            const float sqr_mu2 = s_dot_l2 * s_dot_l2 / sqr_s;
            int skip_mu2 = (sqr_mu2 >= sqr_mumax);
            if (selection.selection_type & RP_SELECTION) {
                const float sqr_rp = (1. - sqr_mu2) * sqr_s;
                if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu2 = 1;
            }
            if (skip_mu && skip_mu2) {
                return;
            }
            s = sqrt(sqr_s);
            if (skip_mu == 0) mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
            if (skip_mu2 == 0) mubin2 = (int) (- s_dot_l2 / s * inv_dmu + mumin_invstep);
        }
        else {
            if (skip_mu) {
                return;
            }
            s = sqrt(sqr_s);
            mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
        }
    }

    float sw = s;
    int kbin = 0;
    if (bin_type == BIN_LIN) {
	kbin = (int) (s*inv_sstep + smin_invstep);
    }
    else {
	for(kbin=nsbin-1;kbin>=1;kbin--) {
	    if(sqr_s >= supp_sqr[kbin-1]) {
		break;
	    }
	}//finding kbin
    }
    kbin *= nmu_bins+1;
    {
	const int ibin = kbin + mubin;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
    }
    if (autocorr == 1) {
	const int ibin = kbin + mubin2;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
    }
}

__global__ void countpairs_s_mu_mocks_pair_weights_kernel_double(double *x0, double *y0, double *z0,
               double *weights0, int numweights0,
               double *x1, double *y1, double *z1, 
               double *weights1, int numweights1,
               int N, int *np0, int *np1, 
               int *same_cell, int64_t *icell0, int64_t *icell1, 
               int *cellpair_lut, int *cellthread_lut,
               int *start_idx0, int *start_idx1,
               double *min_xdiff, double *min_ydiff, 
               double *savg, int *npairs, double *weightavg, const double *supp_sqr,
               const double sqr_smax, const double sqr_smin, const int nsbin,
               const int nmu_bins, 
               const double sqr_mumax, const double inv_dmu, const double mumin_invstep,
               double inv_sstep, double smin_invstep, const selection_struct selection,
               int need_savg, int need_weightavg, int autocorr, int los_type, int bin_type,
               const weight_method_t weight_method, const pair_weight_struct pair_w, double *p_weight, double *p_sep) {
    //thread index tidx 
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= N) return;

    int icellpair = cellpair_lut[blockIdx.x]; //use block index to find cellpair index
    int cell_tidx = cellthread_lut[blockIdx.x] + threadIdx.x; //index within this cellpair from 0 to np0*np1-1

    //icell0, icell1 will translate icellpair to a cell within each lattice
    //start_idx0, start_idx1 then translate to i, j in x0,y0,z0 and x1,y1,z1
    int64_t cellindex0 = icell0[icellpair];
    int64_t cellindex1 = icell1[icellpair];
    //nthreads = np0 * np1 for each cell pair icell
    int this_np0 = np0[cellindex0];
    int this_np1 = np1[cellindex1];
    if (cell_tidx >= this_np0*this_np1) return;

    //start_idx0, start_idx1 give index for first element of x0, y0, z0 and x1, y1, z1 in cell icell.
    //% and / to get i, j
    int i = start_idx0[cellindex0] + cell_tidx / this_np1;
    int j = start_idx1[cellindex1] + cell_tidx % this_np1;

    //get positions for each particle
    double xpos = x0[i];
    double ypos = y0[i];
    double zpos = z0[i];

    double x1pos = x1[j];
    double y1pos = y1[j];
    double z1pos = z1[j];

    if (same_cell[icellpair] && z1pos <= zpos) {
        //return if same particle or in same cell with z1 < z0
        //this way we do not double count pairs
        return;
    }

    const double max_dz = sqrt(sqr_smax - min_xdiff[icellpair]*min_xdiff[icellpair] - min_ydiff[icellpair]*min_ydiff[icellpair]);
    const double this_dz = z1pos-zpos;
    if (abs(this_dz) >= max_dz) {
        //particle too far away in z
        return;
    }

    const double this_dx = x1pos-xpos;
    const double this_dy = y1pos-ypos;
    if ((this_dx*this_dx + this_dy*this_dy + this_dz*this_dz) >= sqr_smax) {
        //particle too far away in separation
        return;
    } 

    //hat1 calcs are done if need_weightavg || ((autocorr == 1) && (los_type == FIRSTPOINT_LOS)
    //need_weightavg is true by definition in this kernel so remove conditional
    double xhat1=NULL, yhat1=NULL, zhat1=NULL;
    const double norm1 = sqrt(x1pos*x1pos + y1pos*y1pos + z1pos*z1pos);
    xhat1 = x1pos/norm1;
    yhat1 = y1pos/norm1;
    zhat1 = z1pos/norm1;

    //hat1 calcs are done if need_weightavg || los_type == FIRSTPOINT_LOS 
    //need_weightavg is true by definition in this kernel so remove conditional
    double xhat0=NULL, yhat0=NULL, zhat0=NULL;
    const double norm0 = sqrt(xpos*xpos + ypos*ypos + zpos*zpos);
    xhat0 = xpos/norm0;
    yhat0 = ypos/norm0;
    zhat0 = zpos/norm0;


    const double parx = xpos + x1pos;
    const double pary = ypos + y1pos; 
    const double parz = zpos + z1pos; 

    const double perpx = x1pos - xpos;
    const double perpy = y1pos - ypos;
    const double perpz = z1pos - zpos;

    //return if > max_dz
    if (perpz > max_dz) {
        return;
    }

    const double sqr_s = perpx*perpx + perpy*perpy + perpz*perpz;
    if(sqr_s >= sqr_smax || sqr_s < sqr_smin) {
        return;
    } 

    double s = 0;
    int mubin = nmu_bins, mubin2 = nmu_bins;
    if (sqr_s <= 0.) {
        mubin2 = mubin = (int) mumin_invstep;
        if((selection.selection_type & RP_SELECTION) && ((0. < selection.rpmin_sqr) || (0. >= selection.rpmax_sqr))) return;
    } 
    else if (los_type == MIDPOINT_LOS) {
        const double s_dot_l = parx*perpx + pary*perpy + parz*perpz;
        const double sqr_l = parx*parx + pary*pary + parz*parz;
        const double sqr_mu = s_dot_l * s_dot_l / (sqr_l * sqr_s);
        if (sqr_mu >= sqr_mumax) {
            return;
        } 
        if (selection.selection_type & RP_SELECTION) {
            const double sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) return;
        }
        const double mu = s_dot_l >= 0 ? sqrt(sqr_mu) : -sqrt(sqr_mu);
        mubin = (int) (mu * inv_dmu + mumin_invstep);
        if (autocorr == 1) mubin2 = (int) (- mu * inv_dmu + mumin_invstep);
        if(need_savg || bin_type == BIN_LIN) s = sqrt(sqr_s);
    }
    else {
        const double s_dot_l = xhat0*perpx + yhat0*perpy + zhat0*perpz;
        const double sqr_mu = s_dot_l * s_dot_l / sqr_s;

        int skip_mu = (sqr_mu >= sqr_mumax);
        if (selection.selection_type & RP_SELECTION) {
            const double sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu = 1;
        }
        if (autocorr == 1) {
            const double s_dot_l2 = xhat1*perpx + yhat1*perpy + zhat1*perpz;
            const double sqr_mu2 = s_dot_l2 * s_dot_l2 / sqr_s;
            int skip_mu2 = (sqr_mu2 >= sqr_mumax);
            if (selection.selection_type & RP_SELECTION) {
                const double sqr_rp = (1. - sqr_mu2) * sqr_s;
                if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu2 = 1;
            }
            if (skip_mu && skip_mu2) {
                return;
            }
            s = sqrt(sqr_s);
            if (skip_mu == 0) mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
            if (skip_mu2 == 0) mubin2 = (int) (- s_dot_l2 / s * inv_dmu + mumin_invstep);
        }
        else {
            if (skip_mu) {
                return;
            }
            s = sqrt(sqr_s);
            mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
        }
    }

    double pairweight = 0; 
    double sw = s;

    //need_weightavg is TRUE so remove conditional and always calculate
    //pairweight - only do simple PAIR_PRODUCT in this kernel
    if (weight_method == PAIR_PRODUCT) pairweight = weights0[i*numweights0] * weights1[j*numweights1];
    else if (weight_method == INVERSE_BITWISE) {
        //use pair_struct and helper method to calculate inverse bitwise weights
        pair_struct_double pair = {.num_weights=numweights0};
        pair.num_integer_weights = numweights1-1;
        for(int w = 0; w < pair.num_weights; w++) {
            pair.weights0[w] = weights0[i*numweights0+w];
            pair.weights1[w] = weights1[j*numweights1+w];
        }
        double pair_costheta_d = xhat1*xhat0 + yhat1*yhat0 + zhat1*zhat0;

        pair.dx = perpx;
        pair.dy = perpy;
        pair.dz = perpz;

        pair.parx = parx;
        pair.pary = pary;
        pair.parz = parz;
        pair.costheta = pair_costheta_d;

        pair.p_weight = p_weight;
        pair.p_sep = p_sep;
        pair.p_num = (int)pair_w.num;
        pair.noffset = pair_w.noffset;
        pair.default_value = (double) pair_w.default_value;

        pairweight = inverse_bitwise_double(&pair);
    }
    if(need_savg) sw = s*pairweight;

    int kbin = 0;
    if (bin_type == BIN_LIN) {
	kbin = (int) (s*inv_sstep + smin_invstep);
    }
    else {
	for(kbin=nsbin-1;kbin>=1;kbin--) {
	    if(sqr_s >= supp_sqr[kbin-1]) {
		break;
	    }
	}//finding kbin
    }
    kbin *= nmu_bins+1;
    {
	const int ibin = kbin + mubin;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
        atomicAdd(&weightavg[ibin], pairweight); //need_weightavg is always true
    }
    if (autocorr == 1) {
	const int ibin = kbin + mubin2;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
        atomicAdd(&weightavg[ibin], pairweight); //need_weightavg is always true
    }
}

__global__ void countpairs_s_mu_mocks_pair_weights_kernel_float(float *x0, float *y0, float *z0,
               float *weights0, int numweights0,
               float *x1, float *y1, float *z1, 
               float *weights1, int numweights1,
               int N, int *np0, int *np1, 
               int *same_cell, int64_t *icell0, int64_t *icell1, 
               int *cellpair_lut, int *cellthread_lut,
               int *start_idx0, int *start_idx1,
               float *min_xdiff, float *min_ydiff, 
               float *savg, int *npairs, float *weightavg, const float *supp_sqr,
               const float sqr_smax, const float sqr_smin, const int nsbin,
               const int nmu_bins, 
               const float sqr_mumax, const float inv_dmu, const float mumin_invstep,
               float inv_sstep, float smin_invstep, const selection_struct selection,
               int need_savg, int need_weightavg, int autocorr, int los_type, int bin_type,
               const weight_method_t weight_method, const pair_weight_struct pair_w, float *p_weight, float *p_sep) {
    //thread index tidx 
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx >= N) return;

    int icellpair = cellpair_lut[blockIdx.x]; //use block index to find cellpair index
    int cell_tidx = cellthread_lut[blockIdx.x] + threadIdx.x; //index within this cellpair from 0 to np0*np1-1

    //icell0, icell1 will translate icellpair to a cell within each lattice
    //start_idx0, start_idx1 then translate to i, j in x0,y0,z0 and x1,y1,z1
    int64_t cellindex0 = icell0[icellpair];
    int64_t cellindex1 = icell1[icellpair];
    //nthreads = np0 * np1 for each cell pair icell
    int this_np0 = np0[cellindex0];
    int this_np1 = np1[cellindex1];
    if (cell_tidx >= this_np0*this_np1) return;

    //start_idx0, start_idx1 give index for first element of x0, y0, z0 and x1, y1, z1 in cell icell.
    //% and / to get i, j
    int i = start_idx0[cellindex0] + cell_tidx / this_np1;
    int j = start_idx1[cellindex1] + cell_tidx % this_np1;

    //get positions for each particle
    float xpos = x0[i];
    float ypos = y0[i];
    float zpos = z0[i];

    float x1pos = x1[j];
    float y1pos = y1[j];
    float z1pos = z1[j];

    if (same_cell[icellpair] && z1pos <= zpos) {
        //return if same particle or in same cell with z1 < z0
        //this way we do not float count pairs
        return;
    }

    const float max_dz = sqrt(sqr_smax - min_xdiff[icellpair]*min_xdiff[icellpair] - min_ydiff[icellpair]*min_ydiff[icellpair]);
    const float this_dz = z1pos-zpos;
    if (abs(this_dz) >= max_dz) {
        //particle too far away in z
        return;
    }

    const float this_dx = x1pos-xpos;
    const float this_dy = y1pos-ypos;
    if ((this_dx*this_dx + this_dy*this_dy + this_dz*this_dz) >= sqr_smax) {
        //particle too far away in separation
        return;
    } 

    //hat1 calcs are done if need_weightavg || ((autocorr == 1) && (los_type == FIRSTPOINT_LOS)
    //need_weightavg is true by definition in this kernel so remove conditional
    float xhat1=NULL, yhat1=NULL, zhat1=NULL;
    const float norm1 = sqrt(x1pos*x1pos + y1pos*y1pos + z1pos*z1pos);
    xhat1 = x1pos/norm1;
    yhat1 = y1pos/norm1;
    zhat1 = z1pos/norm1;

    //hat1 calcs are done if need_weightavg || los_type == FIRSTPOINT_LOS 
    //need_weightavg is true by definition in this kernel so remove conditional
    float xhat0=NULL, yhat0=NULL, zhat0=NULL;
    const float norm0 = sqrt(xpos*xpos + ypos*ypos + zpos*zpos);
    xhat0 = xpos/norm0;
    yhat0 = ypos/norm0;
    zhat0 = zpos/norm0;

    //need_weightavg is TRUE in this kernel BUT pair_costheta_d not used
    //in PAIR_PRODUCT so comment out - will be used for INVERSE_BITWISE 
    //float pair_costheta_d = xhat1*xhat0 + yhat1*yhat0 + zhat1*zhat0;

    const float parx = xpos + x1pos;
    const float pary = ypos + y1pos; 
    const float parz = zpos + z1pos; 

    const float perpx = x1pos - xpos;
    const float perpy = y1pos - ypos;
    const float perpz = z1pos - zpos;

    //return if > max_dz
    if (perpz > max_dz) {
        return;
    }

    const float sqr_s = perpx*perpx + perpy*perpy + perpz*perpz;
    if(sqr_s >= sqr_smax || sqr_s < sqr_smin) {
        return;
    } 

    float s = 0;
    int mubin = nmu_bins, mubin2 = nmu_bins;
    if (sqr_s <= 0.) {
        mubin2 = mubin = (int) mumin_invstep;
        if((selection.selection_type & RP_SELECTION) && ((0. < selection.rpmin_sqr) || (0. >= selection.rpmax_sqr))) return;
    } 
    else if (los_type == MIDPOINT_LOS) {
        const float s_dot_l = parx*perpx + pary*perpy + parz*perpz;
        const float sqr_l = parx*parx + pary*pary + parz*parz;
        const float sqr_mu = s_dot_l * s_dot_l / (sqr_l * sqr_s);
        if (sqr_mu >= sqr_mumax) {
            return;
        } 
        if (selection.selection_type & RP_SELECTION) {
            const float sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) return;
        }
        const float mu = s_dot_l >= 0 ? sqrt(sqr_mu) : -sqrt(sqr_mu);
        mubin = (int) (mu * inv_dmu + mumin_invstep);
        if (autocorr == 1) mubin2 = (int) (- mu * inv_dmu + mumin_invstep);
        if(need_savg || bin_type == BIN_LIN) s = sqrt(sqr_s);
    }
    else {
        const float s_dot_l = xhat0*perpx + yhat0*perpy + zhat0*perpz;
        const float sqr_mu = s_dot_l * s_dot_l / sqr_s;

        int skip_mu = (sqr_mu >= sqr_mumax);
        if (selection.selection_type & RP_SELECTION) {
            const float sqr_rp = (1. - sqr_mu) * sqr_s;
            if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu = 1;
        }
        if (autocorr == 1) {
            const float s_dot_l2 = xhat1*perpx + yhat1*perpy + zhat1*perpz;
            const float sqr_mu2 = s_dot_l2 * s_dot_l2 / sqr_s;
            int skip_mu2 = (sqr_mu2 >= sqr_mumax);
            if (selection.selection_type & RP_SELECTION) {
                const float sqr_rp = (1. - sqr_mu2) * sqr_s;
                if ((sqr_rp < selection.rpmin_sqr) || (sqr_rp >= selection.rpmax_sqr)) skip_mu2 = 1;
            }
            if (skip_mu && skip_mu2) {
                return;
            }
            s = sqrt(sqr_s);
            if (skip_mu == 0) mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
            if (skip_mu2 == 0) mubin2 = (int) (- s_dot_l2 / s * inv_dmu + mumin_invstep);
        }
        else {
            if (skip_mu) {
                return;
            }
            s = sqrt(sqr_s);
            mubin = (int) (s_dot_l / s * inv_dmu + mumin_invstep);
        }
    }

    float pairweight = 0; 
    float sw = s;

    //need_weightavg is TRUE so remove conditional and always calculate
    //pairweight - only do simple PAIR_PRODUCT in this kernel
    if (weight_method == PAIR_PRODUCT) pairweight = weights0[i*numweights0] * weights1[j*numweights1];
    else if (weight_method == INVERSE_BITWISE) {
        //use pair_struct and helper method to calculate inverse bitwise weights
        pair_struct_float pair = {.num_weights=numweights0};
        pair.num_integer_weights = numweights1-1;
        for(int w = 0; w < pair.num_weights; w++) {
            pair.weights0[w] = weights0[i*numweights0+w];
            pair.weights1[w] = weights1[j*numweights1+w];
        }
        float pair_costheta_d = xhat1*xhat0 + yhat1*yhat0 + zhat1*zhat0;

        pair.dx = perpx;
        pair.dy = perpy;
        pair.dz = perpz;

        pair.parx = parx;
        pair.pary = pary;
        pair.parz = parz;
        pair.costheta = pair_costheta_d;

        pair.p_weight = p_weight;
        pair.p_sep = p_sep;
        pair.p_num = (int)pair_w.num;
        pair.noffset = pair_w.noffset;
        pair.default_value = (float) pair_w.default_value;

        pairweight = inverse_bitwise_float(&pair);
    }
    if(need_savg) sw = s*pairweight;

    int kbin = 0;
    if (bin_type == BIN_LIN) {
	kbin = (int) (s*inv_sstep + smin_invstep);
    }
    else {
	for(kbin=nsbin-1;kbin>=1;kbin--) {
	    if(sqr_s >= supp_sqr[kbin-1]) {
		break;
	    }
	}//finding kbin
    }
    kbin *= nmu_bins+1;
    {
	const int ibin = kbin + mubin;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
        atomicAdd(&weightavg[ibin], pairweight); //need_weightavg is always true
    }
    if (autocorr == 1) {
	const int ibin = kbin + mubin2;
        //use atomic add to guarantee atomicity
        atomicAdd(&npairs[ibin], 1);
        if (need_savg) atomicAdd(&savg[ibin], sw);
        atomicAdd(&weightavg[ibin], pairweight); //need_weightavg is always true
    }
}

extern "C" {

//=================== ALLOCATE METHODS =============== //

// ---------- ints ----------

void gpu_allocate_block_luts(int **p_gpu_cellpair_lut, int **p_gpu_cellthread_lut, const int numblocks) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_cellpair_lut), numblocks*sizeof(int));
    hipMallocManaged(&(*p_gpu_cellthread_lut), numblocks*sizeof(int));
}

void gpu_allocate_cell_luts(int **p_gpu_same_cell, int64_t **p_gpu_icell0, int64_t **p_gpu_icell1, const int64_t num_cell_pairs) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_same_cell), num_cell_pairs*sizeof(int));
    hipMallocManaged(&(*p_gpu_icell0), num_cell_pairs*sizeof(int64_t));
    hipMallocManaged(&(*p_gpu_icell1), num_cell_pairs*sizeof(int64_t));
}

void gpu_allocate_lattice_luts(int **p_gpu_np, int **p_gpu_start_idx, const int64_t num_cells) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_np), num_cells*sizeof(int));
    hipMallocManaged(&(*p_gpu_start_idx), num_cells*sizeof(int));
}


// ----------- doubles --------------

void gpu_allocate_mins_double(double **p_gpu_min_dx, double **p_gpu_min_dy, const int64_t num_cell_pairs) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_min_dx), num_cell_pairs*sizeof(double));
    hipMallocManaged(&(*p_gpu_min_dy), num_cell_pairs*sizeof(double));
}   

void gpu_allocate_mocks_double(double **p_X1, double **p_Y1, double **p_Z1, const int64_t ND1) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_X1), ND1*sizeof(double));
    hipMallocManaged(&(*p_Y1), ND1*sizeof(double));
    hipMallocManaged(&(*p_Z1), ND1*sizeof(double));
}

void gpu_allocate_outputs_double(double **p_gpu_savg, int **p_gpu_npairs, const int totnbins) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_savg), totnbins*sizeof(double));
    hipMallocManaged(&(*p_gpu_npairs), totnbins*sizeof(int));
}

void gpu_allocate_one_array_double(double **p_gpu_supp_sqr, const int nsbin) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_supp_sqr), nsbin*sizeof(double));
}

void gpu_allocate_weight_output_double(double **p_gpu_weightavg, const int totnbins) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_weightavg), totnbins*sizeof(double));
}

void gpu_allocate_weights_double(double **p_weights, const int64_t ND1, uint8_t num_weights) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_weights), ND1*num_weights*sizeof(double));
}

// --------------- floats --------------- //

void gpu_allocate_mins_float(float **p_gpu_min_dx, float **p_gpu_min_dy, const int64_t num_cell_pairs) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_min_dx), num_cell_pairs*sizeof(float));
    hipMallocManaged(&(*p_gpu_min_dy), num_cell_pairs*sizeof(float));
}

void gpu_allocate_mocks_float(float **p_X1, float **p_Y1, float **p_Z1, const int64_t ND1) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_X1), ND1*sizeof(float));
    hipMallocManaged(&(*p_Y1), ND1*sizeof(float));
    hipMallocManaged(&(*p_Z1), ND1*sizeof(float));
}

void gpu_allocate_outputs_float(float **p_gpu_savg, int **p_gpu_npairs, const int totnbins) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_savg), totnbins*sizeof(float));
    hipMallocManaged(&(*p_gpu_npairs), totnbins*sizeof(int));
}

void gpu_allocate_one_array_float(float **p_gpu_supp_sqr, const int nsbin) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_supp_sqr), nsbin*sizeof(float));
}

void gpu_allocate_weight_output_float(float **p_gpu_weightavg, const int totnbins) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_gpu_weightavg), totnbins*sizeof(float));
}

void gpu_allocate_weights_float(float **p_weights, const int64_t ND1, uint8_t num_weights) {
    // Allocate Unified Memory – accessible from CPU or GPU
    // Takes pointers as args
    hipMallocManaged(&(*p_weights), ND1*num_weights*sizeof(float));
}

// ============  FREE MEMORY ============= //

// ---------- ints ----------
void gpu_free_block_luts(int *gpu_cellpair_lut, int *gpu_cellthread_lut) {
    hipFree(gpu_cellpair_lut);
    hipFree(gpu_cellthread_lut);
}

void gpu_free_cell_luts(int *gpu_same_cell, int64_t *gpu_icell0, int64_t *gpu_icell1) {
    hipFree(gpu_same_cell);
    hipFree(gpu_icell0);
    hipFree(gpu_icell1);
}

void gpu_free_lattice_luts(int *gpu_np, int *gpu_start_idx) {
    hipFree(gpu_np);
    hipFree(gpu_start_idx);
}

// ----------- doubles --------------

void gpu_free_mins_double(double *gpu_min_dx, double *gpu_min_dy) {
    hipFree(gpu_min_dx);
    hipFree(gpu_min_dy);
}

void gpu_free_mocks_double(double *X1, double *Y1, double *Z1) {
    hipFree(X1);
    hipFree(Y1);
    hipFree(Z1);
}

void gpu_free_outputs_double(double *gpu_savg, int *gpu_npairs) {
    hipFree(gpu_savg);
    hipFree(gpu_npairs);
}

void gpu_free_one_array_double(double *gpu_supp_sqr) {
    hipFree(gpu_supp_sqr);
}

void gpu_free_weight_output_double(double *gpu_weightavg) {
    hipFree(gpu_weightavg);
}

void gpu_free_weights_double(double *weights) {
    hipFree(weights);
}

// --------------- floats --------------- //

void gpu_free_mins_float(float *gpu_min_dx, float *gpu_min_dy) {
    hipFree(gpu_min_dx);
    hipFree(gpu_min_dy);
}

void gpu_free_mocks_float(float *X1, float *Y1, float *Z1) {
    hipFree(X1);
    hipFree(Y1);
    hipFree(Z1);
}

void gpu_free_outputs_float(float *gpu_savg, int *gpu_npairs) {
    hipFree(gpu_savg);
    hipFree(gpu_npairs);
}

void gpu_free_one_array_float(float *gpu_supp_sqr) {
    hipFree(gpu_supp_sqr);
}

void gpu_free_weight_output_float(float *gpu_weightavg) {
    hipFree(gpu_weightavg);
}

void gpu_free_weights_float(float *weights) {
    hipFree(weights);
}

//==========================//


void gpu_device_synchronize() {
  // Wait for GPU to finish before accessing on host
  //This does not need to be called after every kernel invocation,
  //but just before memory is accessed on host
  hipDeviceSynchronize();
}

// =========   Kernel called below ============//

int gpu_batch_countpairs_s_mu_mocks_double(double *x0, double *y0, double *z0,
               double *weights0, uint8_t numweights0,
               double *x1, double *y1, double *z1, 
               double *weights1, uint8_t numweights1,
               const int N, int *np0, int *np1,
               int *same_cell, int64_t *icell0, int64_t *icell1,
               int *cellpair_lut, int *cellthread_lut,
               int *start_idx0, int *start_idx1,
               double *min_xdiff, double *min_ydiff, 
               double *savg, int *npairs, double *weightavg, const double *supp_sqr,
               const double sqr_smax, const double sqr_smin, const int nsbin,
               const int nmu_bins, 
               const double sqr_mumax, const double inv_dmu, const double mumin_invstep,
               double inv_sstep, double smin_invstep, const selection_struct selection,
               int need_savg, const weight_method_t weight_method, const pair_weight_struct pair_weight,
               double *p_weight, double *p_sep,
               int autocorr, int los_type, int bin_type) {
    long threads = N;
    int blocksPerGrid = (threads+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;

    //select kernel based on weight_method - faster to have a base kernel that
    //is not unnecessarily passed extra arrays for weighting calcs that won't
    //be performed

    if (weight_method == NONE) {
        countpairs_s_mu_mocks_kernel_double<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
            x0, y0, z0,
            x1, y1, z1, N,
            np0, np1,
            same_cell, icell0, icell1,
            cellpair_lut, cellthread_lut,
            start_idx0, start_idx1,
            min_xdiff, min_ydiff,
            savg, npairs, supp_sqr,
            sqr_smax, sqr_smin, nsbin, nmu_bins,
            sqr_mumax,inv_dmu,mumin_invstep,
            inv_sstep, smin_invstep, selection,
            need_savg, autocorr, los_type, bin_type);
    } else {
        countpairs_s_mu_mocks_pair_weights_kernel_double<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
            x0, y0, z0, weights0, (int)numweights0,
            x1, y1, z1, weights1, (int)numweights1,
            N, np0, np1,
            same_cell, icell0, icell1,
            cellpair_lut, cellthread_lut,
            start_idx0, start_idx1,
            min_xdiff, min_ydiff, 
            savg, npairs, weightavg, supp_sqr,
            sqr_smax, sqr_smin, nsbin, nmu_bins,
            sqr_mumax,inv_dmu,mumin_invstep,
            inv_sstep, smin_invstep, selection,
            need_savg, 1, autocorr, los_type, bin_type,
            weight_method, pair_weight, p_weight, p_sep);
    }

    //synchronize memory after kernel call
    hipDeviceSynchronize();
//    gpu_print_cuda_error();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) return EXIT_FAILURE;
    return EXIT_SUCCESS;
}

// ----------- float version ----------------

int gpu_batch_countpairs_s_mu_mocks_float(float *x0, float *y0, float *z0,
               float *weights0, uint8_t numweights0,
               float *x1, float *y1, float *z1,
               float *weights1, uint8_t numweights1,
               const int N, int *np0, int *np1,
               int *same_cell, int64_t *icell0, int64_t *icell1,
               int *cellpair_lut, int *cellthread_lut,
               int *start_idx0, int *start_idx1,
               float *min_xdiff, float *min_ydiff,
               float *savg, int *npairs, float *weightavg, const float *supp_sqr,
               const float sqr_smax, const float sqr_smin, const int nsbin,
               const int nmu_bins,
               const float sqr_mumax, const float inv_dmu, const float mumin_invstep,
               float inv_sstep, float smin_invstep, const selection_struct selection,
               int need_savg, const weight_method_t weight_method, const pair_weight_struct pair_weight,
               float *p_weight, float *p_sep,
               int autocorr, int los_type, int bin_type) {
    long threads = N;
    int blocksPerGrid = (threads+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;

    //select kernel based on weight_method - faster to have a base kernel that
    //is not unnecessarily passed extra arrays for weighting calcs that won't
    //be performed

    if (weight_method == NONE) {
        countpairs_s_mu_mocks_kernel_float<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
            x0, y0, z0,
            x1, y1, z1, N,
            np0, np1,
            same_cell, icell0, icell1,
            cellpair_lut, cellthread_lut,
            start_idx0, start_idx1,
            min_xdiff, min_ydiff,
            savg, npairs, supp_sqr,
            sqr_smax, sqr_smin, nsbin, nmu_bins,
            sqr_mumax,inv_dmu,mumin_invstep,
            inv_sstep, smin_invstep, selection,
            need_savg, autocorr, los_type, bin_type);
    } else {
        countpairs_s_mu_mocks_pair_weights_kernel_float<<<blocksPerGrid, THREADS_PER_BLOCK>>>(
            x0, y0, z0, weights0, (int)numweights0,
            x1, y1, z1, weights1, (int)numweights1,
            N, np0, np1,
            same_cell, icell0, icell1,
            cellpair_lut, cellthread_lut,
            start_idx0, start_idx1,
            min_xdiff, min_ydiff,
            savg, npairs, weightavg, supp_sqr,
            sqr_smax, sqr_smin, nsbin, nmu_bins,
            sqr_mumax,inv_dmu,mumin_invstep,
            inv_sstep, smin_invstep, selection,
            need_savg, 1, autocorr, los_type, bin_type,
            weight_method, pair_weight, p_weight, p_sep);
    }

    //synchronize memory after kernel call
    hipDeviceSynchronize();
//    gpu_print_cuda_error();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) return EXIT_FAILURE;
    return EXIT_SUCCESS;
}


void gpu_print_cuda_error() {
       size_t free_byte ;

        size_t total_byte ;

        hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

        if ( hipSuccess != cuda_status ){

            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );

            exit(1);

        }

        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;

        printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

        used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

        hipError_t err = hipGetLastError();
        printf("CUDA Error: %s\n", hipGetErrorString(err));
}

//==============================================
}
